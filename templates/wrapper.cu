#ifndef {{ name|upper }}_WRAPPER_H
#define {{ name|upper }}_WRAPPER_H

#include <cassert>
#include <cstdlib>
#include <cstdio>

#include "{{name}}_particle.h"
#include "{{name}}_tpa_compute_kernel.cu"
#include "{{name}}_bpa_compute_kernel.cu"
#include "decode.cu"

using namespace std;

#define ASSERT_NO_CUDA_ERROR( callReturningErrorstatus ) {     \
  hipError_t err = callReturningErrorstatus;                  \
  if (err != hipSuccess) {                                    \
    fprintf(stderr,                                            \
            "Cuda error (%s/%d) in file '%s' in line %i\n",    \
            hipGetErrorString(err), err, __FILE__, __LINE__); \
    exit(1);                                                   \
  }                                                            \
} while(0);

/*
 * GPU datastructures passed to compute_kernel
 */
static struct particle d_particle_soa;
{% for p in params if not p.is_type('P', 'RO') -%}
  static {{ p.emit_pointer_to_declaration(name_prefix='d_') }};
{% endfor %}
static GpuNeighList *d_nl;

/*
 * Initialize GPU datastructures
 */
void {{name}}_init(int N, int maxpage, int pgsize,
  {% for p in params if p.is_type('-', 'RO') and not p.reload -%}
    {{ p.emit_pointer_to_declaration(name_prefix='h_') }}{{ ',' if not loop.last }}
  {% endfor -%}
) {
  {% for p in params if p.is_type('P', 'RO') -%}
    assert(d_particle_soa.{{ p.name }} == NULL);
  {% endfor %}
  d_nl = new GpuNeighList(N, maxpage, pgsize);
  {% for p in params if not p.is_type('P', 'RO') -%}
    assert({{ p.device_name() }} == NULL);
  {% endfor %}

  {% for p in params if p.is_type('P', 'RO') -%}
    ASSERT_NO_CUDA_ERROR(
      hipMalloc((void **)&d_particle_soa.{{ p.name }}, {{ p.sizeof() }}));
  {% endfor %}
  {% for p in params if not p.is_type('P', 'RO') -%}
    ASSERT_NO_CUDA_ERROR(
      hipMalloc((void **)&{{ p.device_name() }}, {{ p.sizeof() }}));
  {% endfor %}

  {% for p in params if p.is_type('-', 'RO') and not p.reload -%}
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_particle_soa.{{ p.name }}, {{ p.emit_name(name_prefix='h_') }}, {{ p.sizeof() }}, hipMemcpyHostToDevice));
  {% endfor -%}
}

/*
 * (Re)fill neighbor list
 */
void {{name}}_update_neigh(
  int *h_numneigh,
  int **h_firstneigh,
  int **h_pages,
  {% for p in params if p.is_type('N', '-') -%} 
    {{ p.type }} **{{ p.emit_name(name_prefix='h_') }}{{ ',' if not loop.last }}
  {% endfor -%}
) {
  d_nl->reload(h_numneigh, h_firstneigh, h_pages);
  {% for p in params if p.is_type('N', '-') -%} 
    d_nl->load_pages<{{ p.type }}>({{ p.device_name() }}, {{ p.emit_name(name_prefix='h_') }}, {{ p.arity }});
  {% endfor -%}
}

void {{name}}_exit() {
  {% for p in params if p.is_type('P', 'RO') -%}
    assert(d_particle_soa.{{ p.name }});
  {% endfor %}
  assert(d_nl);
  {% for p in params if not p.is_type('P', 'RO') -%}
    assert({{ p.device_name() }});
  {% endfor %}

  {% for p in params if p.is_type('P', 'RO') -%}
    hipFree(d_particle_soa.{{ p.name }});
  {% endfor %}
  delete(d_nl);
  {% for p in params if not p.is_type('P', 'RO') -%}
    hipFree({{ p.device_name() }});
  {% endfor %}
}

void {{name}}_run(int N, int NSLOT, int maxpage, int pgsize,
  {% for p in params -%}
    {%- if p.is_type('-', 'RO') and p.reload -%}
      {{ p.emit_pointer_to_declaration(name_prefix='h_') }}{{ ',' if not loop.last }}
    {%- elif p.is_type('-', 'RW') or p.is_type('-', 'SUM') -%}
      {{ p.emit_pointer_to_declaration(name_prefix='h_') }}{{ ',' if not loop.last }}
    {%- else -%}
      //{{ p.name }}
    {%- endif %}
  {% endfor %}
  ) {

  {% for p in params if p.is_type('-', 'RO') and p.reload -%}
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_particle_soa.{{ p.name }}, {{ p.emit_name(name_prefix='h_') }}, {{ p.sizeof() }}, hipMemcpyHostToDevice));
  {% endfor %}
  {% for p in params if p.is_type('P', 'RW') or p.is_type('P', 'SUM') -%}
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy({{ p.device_name() }}, {{ p.emit_name(name_prefix='h_') }}, {{ p.sizeof() }}, hipMemcpyHostToDevice));
  {% endfor %}

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Pre-compute-kernel error: %s.\n", hipGetErrorString(err));
    exit(1);
  }
#ifdef COMPUTE_TPA
  const int blockSize = 128;
  dim3 gridSize((N / blockSize)+1);
  {{name}}_tpa_compute_kernel<<<gridSize, blockSize>>>(
    N, d_particle_soa, 
    d_nl->d_numneigh, d_nl->d_pageidx, d_nl->d_offset, d_nl->pgsize, d_nl->d_neighidx,
    {% for p in params if not p.is_type('P', 'RO') -%}
      {{ p.device_name() }}{{ ',' if not loop.last }}
    {% endfor %}
  );
#else
  const int blockSize = NSLOT;
  dim3 gridSize(N);
  size_t sharedMemSize = 0;
  {% for p in params if p.is_type('P', 'SUM') -%}
    sharedMemSize += NSLOT * {{ p.arity }} * {{ p.sizeof_in_chars() }}; // {{ p.device_name() }}
  {% endfor %}
  {{name}}_bpa_compute_kernel<<<gridSize, blockSize, sharedMemSize>>>(
    N, NSLOT, d_particle_soa,
    d_nl->d_numneigh, d_nl->d_pageidx, d_nl->d_offset, d_nl->pgsize, d_nl->d_neighidx,
    {% for p in params if not p.is_type('P', 'RO') -%}
      {{ p.device_name() }}{{ ',' if not loop.last }}
    {% endfor %}
  );
#endif
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Post-compute-kernel error: %s.\n", hipGetErrorString(err));
    exit(1);
  }

  {% for p in params if p.is_type('P', 'RW') or p.is_type('P', 'SUM') -%}
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy({{ p.emit_name(name_prefix='h_') }}, {{ p.device_name() }}, {{ p.sizeof() }}, hipMemcpyDeviceToHost));
  {% endfor %}
}

#endif
